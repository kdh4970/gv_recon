#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
  Marching cubes

  This sample extracts a geometric isosurface from a volume dataset using
  the marching cubes algorithm. It uses the scan (prefix sum) function from
  the Thrust library to perform stream compaction.  Similar techniques can
  be used for other problems that require a variable-sized output per
  thread.

  For more information on marching cubes see:
  http://local.wasp.uwa.edu.au/~pbourke/geometry/polygonise/
  http://en.wikipedia.org/wiki/Marching_cubes

  Volume data courtesy:
  http://www9.informatik.uni-erlangen.de/External/vollib/

  For more information on the Thrust library
  http://code.google.com/p/thrust/

  The algorithm consists of several stages:

  1. Execute "classifyVoxel" kernel
  This evaluates the volume at the corners of each voxel and computes the
  number of vertices each voxel will generate.
  It is executed using one thread per voxel.
  It writes two arrays - voxelOccupied and voxelVertices to global memory.
  voxelOccupied is a flag indicating if the voxel is non-empty.

  2. Scan "voxelOccupied" array (using Thrust scan)
  Read back the total number of occupied voxels from GPU to CPU.
  This is the sum of the last value of the exclusive scan and the last
  input value.

  3. Execute "compactVoxels" kernel
  This compacts the voxelOccupied array to get rid of empty voxels.
  This allows us to run the complex "generateTriangles" kernel on only
  the occupied voxels.

  4. Scan voxelVertices array
  This gives the start address for the vertex data for each voxel.
  We read back the total number of vertices generated from GPU to CPU.

  Note that by using a custom scan function we could combine the above two
  scan operations above into a single operation.

  5. Execute "generateTriangles" kernel
  This runs only on the occupied voxels.
  It looks up the field values again and generates the triangle data,
  using the results of the scan to write the output to the correct addresses.
  The marching cubes look-up tables are stored in 1D textures.

  6. Render geometry
  Using number of vertices from readback.
*/
#ifndef __MC_CU__
#define __MC_CU__
#include "marchingCubes.h"





void dumpFile(void *dData, int data_bytes, const char *file_name) {
  void *hData = malloc(data_bytes);
  checkCudaErrors(hipMemcpy(hData, dData, data_bytes, hipMemcpyDeviceToHost));
  sdkDumpBin(hData, data_bytes, file_name);
  free(hData);
}

template <class T>
void dumpBuffer(T *d_buffer, int nelements, int size_element) {
  uint bytes = nelements * size_element;
  T *h_buffer = (T *)malloc(bytes);
  checkCudaErrors(
      hipMemcpy(h_buffer, d_buffer, bytes, hipMemcpyDeviceToHost));

  for (int i = 0; i < nelements; i++) {
    printf("%d: %u\n", i, h_buffer[i]);
  }

  printf("\n");
  free(h_buffer);
}


void runAutoTest(int argc, char **argv) {
  findCudaDevice(argc, (const char **)argv);

  // Initialize CUDA buffers for Marching Cubes
  initMC(argc, argv);

  computeIsosurface();

  char *ref_file = NULL;
  getCmdLineArgumentString(argc, (const char **)argv, "file", &ref_file);

  enum DUMP_TYPE { DUMP_POS = 0, DUMP_NORMAL, DUMP_VOXEL };
  int dump_option = getCmdLineArgumentInt(argc, (const char **)argv, "dump");

  bool bTestResult = true;

  switch (dump_option) {
    case DUMP_POS:
      dumpFile((void *)d_pos, sizeof(float4) * maxVerts,
              "marchCube_posArray.bin");
      bTestResult = sdkCompareBin2BinFloat(
          "marchCube_posArray.bin", "posArray.bin",
          maxVerts * sizeof(float) * 4, EPSILON, THRESHOLD, argv[0]);
      break;

    case DUMP_NORMAL:
      dumpFile((void *)d_normal, sizeof(float4) * maxVerts,
              "marchCube_normalArray.bin");
      bTestResult = sdkCompareBin2BinFloat(
          "marchCube_normalArray.bin", "normalArray.bin",
          maxVerts * sizeof(float) * 4, EPSILON, THRESHOLD, argv[0]);
      break;

    case DUMP_VOXEL:
      dumpFile((void *)d_compVoxelArray, sizeof(uint) * numVoxels,
              "marchCube_compVoxelArray.bin");
      bTestResult = sdkCompareBin2BinFloat(
          "marchCube_compVoxelArray.bin", "compVoxelArray.bin",
          numVoxels * sizeof(uint), EPSILON, THRESHOLD, argv[0]);
      break;

    default:
      printf("Invalid validation flag!\n");
      printf("-dump=0 <check position>\n");
      printf("-dump=1 <check normal>\n");
      printf("-dump=2 <check voxel>\n");
      exit(EXIT_SUCCESS);
  }

  exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

////////////////////////////////////////////////////////////////////////////////
// initialize marching cubes
////////////////////////////////////////////////////////////////////////////////
void initMC(int argc, char **argv) {
  // parse command line arguments
  int n;
  if (checkCmdLineFlag(argc, (const char **)argv, "grid")) {
    n = getCmdLineArgumentInt(argc, (const char **)argv, "grid");
    gridSizeLog2.x = gridSizeLog2.y = gridSizeLog2.z = n;
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "gridx")) {
    n = getCmdLineArgumentInt(argc, (const char **)argv, "gridx");
    gridSizeLog2.x = n;
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "gridy")) {
    n = getCmdLineArgumentInt(argc, (const char **)argv, "gridy");
    gridSizeLog2.y = n;
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "gridz")) {
    n = getCmdLineArgumentInt(argc, (const char **)argv, "gridz");
    gridSizeLog2.z = n;
  }

  char *filename;

  if (getCmdLineArgumentString(argc, (const char **)argv, "file", &filename)) {
    volumeFilename = filename;
  }

  gridSize =
      make_uint3(1 << gridSizeLog2.x, 1 << gridSizeLog2.y, 1 << gridSizeLog2.z);
  gridSizeMask = make_uint3(gridSize.x - 1, gridSize.y - 1, gridSize.z - 1);
  gridSizeShift =
      make_uint3(0, gridSizeLog2.x, gridSizeLog2.x + gridSizeLog2.y);

  numVoxels = gridSize.x * gridSize.y * gridSize.z;
  voxelSize =
      make_float3(2.0f / gridSize.x, 2.0f / gridSize.y, 2.0f / gridSize.z);
  maxVerts = gridSize.x * gridSize.y * 100;

  printf("grid: %d x %d x %d = %d voxels\n", gridSize.x, gridSize.y, gridSize.z,
        numVoxels);
  printf("max verts = %d\n", maxVerts);

#if SAMPLE_VOLUME
  // load volume data
  char *path = sdkFindFilePath(volumeFilename, argv[0]);

  if (path == NULL) {
    fprintf(stderr, "Error finding file '%s'\n", volumeFilename);

    exit(EXIT_FAILURE);
  }

  int size = gridSize.x * gridSize.y * gridSize.z * sizeof(uchar);
  uchar *volume = loadRawFile(path, size);
  checkCudaErrors(hipMalloc((void **)&d_volume, size));
  checkCudaErrors(hipMemcpy(d_volume, volume, size, hipMemcpyHostToDevice));
  free(volume);

  createVolumeTexture(d_volume, size);
#endif

  if (g_bValidate) {
    hipMalloc((void **)&(d_pos), maxVerts * sizeof(float) * 4);
    hipMalloc((void **)&(d_normal), maxVerts * sizeof(float) * 4);
  } else {
    // create VBOs
    createVBO(&posVbo, maxVerts * sizeof(float) * 4);
    // DEPRECATED: checkCudaErrors( cudaGLRegisterBufferObject(posVbo) );
    checkCudaErrors(hipGraphicsGLRegisterBuffer(
        &cuda_posvbo_resource, posVbo, cudaGraphicsMapFlagsWriteDiscard));

    createVBO(&normalVbo, maxVerts * sizeof(float) * 4);
    // DEPRECATED: checkCudaErrors(cudaGLRegisterBufferObject(normalVbo));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(
        &cuda_normalvbo_resource, normalVbo, cudaGraphicsMapFlagsWriteDiscard));
  }
  // allocate textures
  allocateTextures(&d_edgeTable, &d_triTable, &d_numVertsTable);

  // allocate device memory
  unsigned int memSize = sizeof(uint) * numVoxels;
  checkCudaErrors(hipMalloc((void **)&d_voxelVerts, memSize));
  checkCudaErrors(hipMalloc((void **)&d_voxelVertsScan, memSize));
  checkCudaErrors(hipMalloc((void **)&d_voxelOccupied, memSize));
  checkCudaErrors(hipMalloc((void **)&d_voxelOccupiedScan, memSize));
  checkCudaErrors(hipMalloc((void **)&d_compVoxelArray, memSize));
}

void cleanup() {
  if (g_bValidate) {
    hipFree(d_pos);
    hipFree(d_normal);
  } else {
    sdkDeleteTimer(&timer);

    deleteVBO(&posVbo, &cuda_posvbo_resource);
    deleteVBO(&normalVbo, &cuda_normalvbo_resource);
  }
  destroyAllTextureObjects();
  checkCudaErrors(hipFree(d_edgeTable));
  checkCudaErrors(hipFree(d_triTable));
  checkCudaErrors(hipFree(d_numVertsTable));

  checkCudaErrors(hipFree(d_voxelVerts));
  checkCudaErrors(hipFree(d_voxelVertsScan));
  checkCudaErrors(hipFree(d_voxelOccupied));
  checkCudaErrors(hipFree(d_voxelOccupiedScan));
  checkCudaErrors(hipFree(d_compVoxelArray));

  if (d_volume) {
    checkCudaErrors(hipFree(d_volume));
  }
}


void runGraphicsTest(int argc, char **argv) {
  printf("MarchingCubes\n");

  if (checkCmdLineFlag(argc, (const char **)argv, "device")) {
    printf("[%s]\n", argv[0]);
    printf("   Does not explicitly support -device=n in OpenGL mode\n");
    printf("   To use -device=n, the sample must be running w/o OpenGL\n\n");
    printf(" > %s -device=n -file=<reference> -dump=<0/1/2>\n", argv[0]);
    exit(EXIT_SUCCESS);
  }

  // First initialize OpenGL context, so we can properly set the GL for CUDA.
  // This is necessary in order to achieve optimal performance with OpenGL/CUDA
  // interop.
  if (false == initGL(&argc, argv)) {
    return;
  }

  findCudaDevice(argc, (const char **)argv);

  // register callbacks
  // glutDisplayFunc(display);
  glutKeyboardFunc(keyboard);
  glutMouseFunc(mouse);
  glutMotionFunc(motion);
  glutReshapeFunc(reshape);
  glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
  initMenus();

  // Initialize CUDA buffers for Marching Cubes
  initMC(argc, argv);

  sdkCreateTimer(&timer);

  // start rendering mainloop
  glutMainLoop();
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void computeIsosurface() {
  int threads = 128;
  dim3 grid(numVoxels / threads, 1, 1);

  // get around maximum grid size of 65535 in each dimension
  if (grid.x > 65535) {
    grid.y = grid.x / 32768;
    grid.x = 32768;
  }

  // calculate number of vertices need per voxel
  launch_classifyVoxel(grid, threads, d_voxelVerts, d_voxelOccupied, d_volume,
                      gridSize, gridSizeShift, gridSizeMask, numVoxels,
                      voxelSize, isoValue);
#if DEBUG_BUFFERS
  printf("voxelVerts:\n");
  dumpBuffer(d_voxelVerts, numVoxels, sizeof(uint));
#endif

#if SKIP_EMPTY_VOXELS
  // scan voxel occupied array
  ThrustScanWrapper(d_voxelOccupiedScan, d_voxelOccupied, numVoxels);

#if DEBUG_BUFFERS
  printf("voxelOccupiedScan:\n");
  dumpBuffer(d_voxelOccupiedScan, numVoxels, sizeof(uint));
#endif

  // read back values to calculate total number of non-empty voxels
  // since we are using an exclusive scan, the total is the last value of
  // the scan result plus the last value in the input array
  {
    uint lastElement, lastScanElement;
    checkCudaErrors(hipMemcpy((void *)&lastElement,
                              (void *)(d_voxelOccupied + numVoxels - 1),
                              sizeof(uint), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy((void *)&lastScanElement,
                              (void *)(d_voxelOccupiedScan + numVoxels - 1),
                              sizeof(uint), hipMemcpyDeviceToHost));
    activeVoxels = lastElement + lastScanElement;
  }

  if (activeVoxels == 0) {
    // return if there are no full voxels
    totalVerts = 0;
    return;
  }

  // compact voxel index array
  launch_compactVoxels(grid, threads, d_compVoxelArray, d_voxelOccupied,
                      d_voxelOccupiedScan, numVoxels);
  getLastCudaError("compactVoxels failed");

#endif  // SKIP_EMPTY_VOXELS

  // scan voxel vertex count array
  ThrustScanWrapper(d_voxelVertsScan, d_voxelVerts, numVoxels);

#if DEBUG_BUFFERS
  printf("voxelVertsScan:\n");
  dumpBuffer(d_voxelVertsScan, numVoxels, sizeof(uint));
#endif

  // readback total number of vertices
  {
    uint lastElement, lastScanElement;
    checkCudaErrors(hipMemcpy((void *)&lastElement,
                              (void *)(d_voxelVerts + numVoxels - 1),
                              sizeof(uint), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy((void *)&lastScanElement,
                              (void *)(d_voxelVertsScan + numVoxels - 1),
                              sizeof(uint), hipMemcpyDeviceToHost));
    totalVerts = lastElement + lastScanElement;
  }

  // generate triangles, writing to vertex buffers
  if (!g_bValidate) {
    size_t num_bytes;
    // DEPRECATED: checkCudaErrors(cudaGLMapBufferObject((void**)&d_pos,
    // posVbo));
    checkCudaErrors(hipGraphicsMapResources(1, &cuda_posvbo_resource, 0));
    checkCudaErrors(hipGraphicsResourceGetMappedPointer(
        (void **)&d_pos, &num_bytes, cuda_posvbo_resource));

    // DEPRECATED: checkCudaErrors(cudaGLMapBufferObject((void**)&d_normal,
    // normalVbo));
    checkCudaErrors(hipGraphicsMapResources(1, &cuda_normalvbo_resource, 0));
    checkCudaErrors(hipGraphicsResourceGetMappedPointer(
        (void **)&d_normal, &num_bytes, cuda_normalvbo_resource));
  }

#if SKIP_EMPTY_VOXELS
  dim3 grid2((int)ceil(activeVoxels / (float)NTHREADS), 1, 1);
#else
  dim3 grid2((int)ceil(numVoxels / (float)NTHREADS), 1, 1);
#endif

  while (grid2.x > 65535) {
    grid2.x /= 2;
    grid2.y *= 2;
  }

#if SAMPLE_VOLUME
  launch_generateTriangles2(grid2, NTHREADS, d_pos, d_normal, d_compVoxelArray,
                            d_voxelVertsScan, d_volume, gridSize, gridSizeShift,
                            gridSizeMask, voxelSize, isoValue, activeVoxels,
                            maxVerts);
#else
  launch_generateTriangles(grid2, NTHREADS, d_pos, d_normal, d_compVoxelArray,
                          d_voxelVertsScan, gridSize, gridSizeShift,
                          gridSizeMask, voxelSize, isoValue, activeVoxels,
                          maxVerts);
#endif

  if (!g_bValidate) {
    // DEPRECATED:      checkCudaErrors(cudaGLUnmapBufferObject(normalVbo));
    checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_normalvbo_resource, 0));
    // DEPRECATED:      checkCudaErrors(cudaGLUnmapBufferObject(posVbo));
    checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_posvbo_resource, 0));
  }
}








////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/) {
  switch (key) {
    case (27):
      cleanup();
      exit(EXIT_SUCCESS);

    case '=':
      isoValue += 0.01f;
      break;

    case '-':
      isoValue -= 0.01f;
      break;

    case '+':
      isoValue += 0.1f;
      break;

    case '_':
      isoValue -= 0.1f;
      break;

    case 'w':
      wireframe = !wireframe;
      break;

    case ' ':
      animate = !animate;
      break;

    case 'l':
      lighting = !lighting;
      break;

    case 'r':
      render = !render;
      break;

    case 'c':
      compute = !compute;
      break;
  }

  printf("isoValue = %f\n", isoValue);
  printf("voxels = %d\n", activeVoxels);
  printf("verts = %d\n", totalVerts);
  printf("occupancy: %d / %d = %.2f%%\n", activeVoxels, numVoxels,
        activeVoxels * 100.0f / (float)numVoxels);

  if (!compute) {
    computeIsosurface();
  }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y) {
  if (state == GLUT_DOWN) {
    mouse_buttons |= 1 << button;
  } else if (state == GLUT_UP) {
    mouse_buttons = 0;
  }

  mouse_old_x = x;
  mouse_old_y = y;
}

void motion(int x, int y) {
  float dx = (float)(x - mouse_old_x);
  float dy = (float)(y - mouse_old_y);

  if (mouse_buttons == 1) {
    rotate.x += dy * 0.2f;
    rotate.y += dx * 0.2f;
  } else if (mouse_buttons == 2) {
    translate.x += dx * 0.01f;
    translate.y -= dy * 0.01f;
  } else if (mouse_buttons == 3) {
    translate.z += dy * 0.01f;
  }

  mouse_old_x = x;
  mouse_old_y = y;
  glutPostRedisplay();
}

void idle() {
  animation();
  glutPostRedisplay();
}

void reshape(int w, int h) {
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  gluPerspective(60.0, (float)w / (float)h, 0.1, 10.0);

  glMatrixMode(GL_MODELVIEW);
  glViewport(0, 0, w, h);
}

void mainMenu(int i) { keyboard((unsigned char)i, 0, 0); }

#endif